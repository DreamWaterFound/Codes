// 这个程序选中符合指定条件的设备

// CUDA 支持
#include "hip/hip_runtime.h"


// 传统 C++ 支持
#include <iostream>
 
using namespace std;
 
// 主函数，还没有输入参数
int main()
{
	//定义需要的设备属性
	hipDeviceProp_t devicePropDefined;
    memset(&devicePropDefined, 0, sizeof(hipDeviceProp_t));  //设置devicepropDefined的值
    // 版本号的要求
	devicePropDefined.major = 5;
	devicePropDefined.minor = 2;
 
	int devicedChoosed;  //选中的设备ID
	hipError_t hipError_t;
	hipGetDevice(&devicedChoosed);  //获取当前设备ID
	cout << "当前使用设备的编号： " << devicedChoosed << endl;
 
	hipChooseDevice(&devicedChoosed, &devicePropDefined);  //查找符合要求的设备ID
	cout << "满足指定属性要求的设备的编号： " << devicedChoosed << endl;
 
	hipError_t = hipSetDevice(devicedChoosed); //设置选中的设备为下文的运行设备
 
	if (hipError_t == hipSuccess)
		cout << "设备选取成功！" << endl;
	else
        cout << "设备选取失败！" << endl;
        
    char c;
    cin>>c;
	return 0;
}
